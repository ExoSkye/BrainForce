#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <ctime>

#include "adler.cuh"

const uint64_t LEN = 4;

uint32_t adler_cpu(const unsigned char* inp_str, const size_t inp_len) {
    uint16_t a = 1;
    uint16_t b = 0;

    for (int i = 0; i < inp_len; i++) {
        const unsigned char inp_char = inp_str[i];
        a += (uint8_t) inp_char;
        a %= 65521;

        b += a;
        b %= 65521;
    }

    uint32_t out = b << 16 | a;

    return out;
}

int main() {
    auto** bruteforce_bytes = (unsigned char**)malloc(sizeof(unsigned char*) * 65536);
    auto* zero_bytes = (unsigned char*)calloc(sizeof(unsigned char), LEN);

    unsigned char** d_bruteforce_bytes = nullptr;

    hipError_t err;

    err = hipMalloc((void**)&d_bruteforce_bytes, sizeof(unsigned char*) * 65536);
    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(1);
    }

    for (int i = 0; i < 65536; i++) {
        err = hipMalloc((void**)&bruteforce_bytes[i], sizeof(unsigned char) * LEN);
        if (err != hipSuccess) {
            printf("Failed to initialize, exiting...\n");
            exit(1);
        }
        err = hipMemcpy(bruteforce_bytes[i], zero_bytes, sizeof(unsigned char) * LEN, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("Failed to initialize, exiting...\n");
            exit(1);
        }
    }

    err = hipMemcpy(d_bruteforce_bytes, bruteforce_bytes, sizeof(unsigned char*) * 65536,
                           hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(1);
    }

    auto* matches = (bool*)calloc(sizeof(bool), 65536);

    bool* d_matches = NULL;

    err = hipMalloc((void**)&d_matches, sizeof(bool) * 65536);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(0);
    }

    err = hipMemcpy(d_matches, matches, sizeof(bool) * 65536, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(0);
    }

    const uint32_t target = adler_cpu((const unsigned char*)("a"), 1);

    bool found = false;

    for (int len = 1; len < LEN + 1 && !found; len++) {
        printf("Trying length: %i\n", len);

        int iter = 0;

        double possible_combos = pow(2, 8 * len);

        printf("%f possible combinations\n", possible_combos);

        uint64_t combos = 0;

        time_t start_time = time(NULL);

        while (!found) {
            adler<<<256, 256>>>(d_matches, target, d_bruteforce_bytes, len);
            increment<<<256, 256>>>(d_bruteforce_bytes, len);

            if (iter % 10000 == 5000 || combos == possible_combos) {
                time_t cur_time = time(NULL) - start_time;
                time_t time_left = ((double) cur_time / (double) combos) * (possible_combos - (double) combos);
                char* time_str = (char*) calloc(sizeof(char), 10);
                struct tm* timeinfo = localtime(&time_left);

                strftime(time_str, 10, "%X", timeinfo);
                printf("Checking results (%f%% done - estimated time: %s)\n", (double) combos / possible_combos * 100,
                       time_str);
                hipMemcpy(matches, d_matches, sizeof(bool) * 65536, hipMemcpyDeviceToHost);

                for (int i = 0; i < 65536; i++) {
                    if (matches[i]) {
                        printf("FOUND MATCH ON THREAD %d\n", i);

                        hipMemcpy(zero_bytes,
                                   bruteforce_bytes[i],
                                   sizeof(unsigned char) * len,
                                   hipMemcpyDeviceToHost);

                        unsigned char* cpy_bytes = zero_bytes;

                        while(*cpy_bytes)
                            printf("%02x", (unsigned int) *cpy_bytes++);
                        printf("\n");

                        found = true;
                        break;
                    }
                }
            }

            iter++;


            if ((double) combos > possible_combos) {
                printf("Couldn't find that value in given length space, maybe try a different length?\n");
                break;
            }

            combos += std::min(pow(2, 8 * len), 65536.0);
        }
    }

    for (int i = 0; i < 65536; i++) {
        hipFree(bruteforce_bytes[i]);
    }

    hipFree(d_bruteforce_bytes);
    hipFree(d_matches);

    free(bruteforce_bytes);
    free(matches);
    free(zero_bytes);

    return 0;
}
