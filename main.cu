#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <ctime>
#include <cinttypes>
#include <fstream>

#include "fnv1a.cuh"
#include <boost/program_options.hpp>
#include <iostream>

//const uint64_t LEN = 5;

namespace po = boost::program_options;

int main(int argc, char** argv) {

    po::options_description desc("Parameters");
    desc.add_options()
        ("help", "Print a help message")
        ("target", po::value<std::string>(), "The hash to find (required)")
        ("length", po::value<uint64_t>(), "The max length of the string the hash can represent (required)")
        ("precompute", po::value<float>(), "Number of gigabytes of bruteforce inputs to precompute (required)")
    ;

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << std::endl;
    }

    if (!vm.count("target") || !vm.count("length") || !vm.count("precompute")) {
        std::cout << "Required parameters are not set, exiting..." << std::endl;
        exit(1);
    }

    uint64_t LEN = vm["length"].as<uint64_t>();
    std::string target_std_string = vm["target"].as<std::string>();
    const char* target_str = target_std_string.c_str();
    uint64_t precompute = vm["precompute"].as<float>() * 1073741824;

    auto** bruteforce_bytes = (unsigned char**)malloc(sizeof(unsigned char*) * 65536);
    auto* zero_bytes = (unsigned char*)calloc(sizeof(unsigned char), LEN);

    unsigned char** d_bruteforce_bytes = nullptr;

    hipError_t err;

    err = hipMalloc((void**)&d_bruteforce_bytes, sizeof(unsigned char*) * 65536);
    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(1);
    }

    for (int i = 0; i < 65536; i++) {
        err = hipMalloc((void**)&bruteforce_bytes[i], sizeof(unsigned char) * LEN);
        if (err != hipSuccess) {
            printf("Failed to initialize, exiting...\n");
            exit(1);
        }
        err = hipMemcpy(bruteforce_bytes[i], zero_bytes, sizeof(unsigned char) * LEN, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("Failed to initialize, exiting...\n");
            exit(1);
        }
    }

    err = hipMemcpy(d_bruteforce_bytes, bruteforce_bytes, sizeof(unsigned char*) * 65536,
                           hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(1);
    }

    auto* matches = (bool*)calloc(sizeof(bool), 65536);

    bool* d_matches = NULL;

    err = hipMalloc((void**)&d_matches, sizeof(bool) * 65536);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(0);
    }

    err = hipMemcpy(d_matches, matches, sizeof(bool) * 65536, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        printf("Failed to initialize, exiting...\n");
        exit(0);
    }

    const __uint128_t target = fnv1a64_cpu((const unsigned char*)target_str, LEN);

    bool found = false;

    char* time_str = (char*) calloc(sizeof(char), 10);

    for (int len = 1; len < LEN + 1 && !found; len++) {
        printf("Trying length: %i\n", len);

        int iter = 0;

        double possible_combos = pow(2, 8 * len);

        printf("%f possible combinations\n", possible_combos);

        uint64_t combos = 0;

        time_t start_time = time(NULL);

        while (!found) {
            fnv1a64<<<256, 256>>>(d_matches, target, d_bruteforce_bytes, len);
            
            if (iter % 10000 == 5000 || combos == possible_combos) {
                time_t cur_time = time(NULL) - start_time;
                time_t time_left = ((double) cur_time / (double) combos) * (possible_combos - (double) combos);
                struct tm* timeinfo = localtime(&time_left);

                strftime(time_str, 10, "%X", timeinfo);
                printf("Checking results (%f%% done - estimated time: %s - %f H/s)\n", (double) combos /
                    possible_combos * 100, time_str, (float) combos / (float) cur_time);
                hipMemcpy(matches, d_matches, sizeof(bool) * 65536, hipMemcpyDeviceToHost);

                for (int i = 0; i < 65536; i++) {
                    if (matches[i]) {
                        printf("FOUND MATCH ON THREAD %d\n", i);

                        hipMemcpy(zero_bytes,
                                   bruteforce_bytes[i],
                                   sizeof(unsigned char) * len,
                                   hipMemcpyDeviceToHost);

                        unsigned char* cpy_bytes = zero_bytes;

                        while(cpy_bytes < zero_bytes + LEN)
                            printf("%02x", (unsigned int) *cpy_bytes++);
                        printf("\n");

                        found = true;
                        break;
                    }
                }
            }

            iter++;


            if ((double) combos > possible_combos) {
                printf("Couldn't find that value in given length space, maybe try a different length?\n");
                break;
            }

            combos += std::min(pow(2, 8 * len), 65536.0);
        }
    }

    for (int i = 0; i < 65536; i++) {
        hipFree(bruteforce_bytes[i]);
    }

    hipFree(d_bruteforce_bytes);
    hipFree(d_matches);

    free(bruteforce_bytes);
    free(matches);
    free(zero_bytes);
    free(time_str);

    return 0;
}
